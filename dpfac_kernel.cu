#include "hip/hip_runtime.h"
//
//  dpfac_kernel.cu
//  PFAC-Duo
//
//  Created by CbS Ghost on 2017/2/10.
//  Copyright (c) 2017 UrBX Creative Studio. All rights reserved.
//

#include "dpfac_kernel.cuh"
static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

typedef struct _HashCell {
    uint16_t state;
    uint16_t next_state;
} HashCell __attribute__ ((aligned (4)));

__global__ void DpfacKernelCU_Normal(int32_t *result, uint16_t *input_buf, const uint32_t input_buf_size, const __restrict__ int32_t *hash_info, const __restrict__ HashCell *hash_data, const uint32_t hash_data_size, const uint16_t pat_count, const uint16_t max_pat_block_len)
{
    uint32_t gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= input_buf_size) {
        return;
    }
    
    uint32_t hash_data_size_reg = hash_data_size;
    uint16_t pat_count_reg = pat_count;
    
    int32_t  pos[2];
    int32_t  run_nextstate[2] = {0, 0};
    uint16_t in_pat, pat_key[2];
    uint16_t match_result[2];
    HashCell h_cell[2] = {{0xFFFE, 0}, {0xFFFE, 0}};
    
    // match the first ushort
    pat_key[0] = input_buf[gid];
    pat_key[1] = pat_key[0] & 0xFF00;
    pos[0] = hash_info[0] + pat_key[0];
    pos[1] = hash_info[0] + pat_key[1];
    h_cell[0] = (pos[0] >= 0 && pos[0] < (int32_t)hash_data_size_reg && (pat_key[0] & 0x00FF)) ? hash_data[pos[0]] : h_cell[0];
    h_cell[1] = (pos[1] >= 0 && pos[1] < (int32_t)hash_data_size_reg) ? hash_data[pos[1]] : h_cell[1];
    #pragma unroll
    for (int32_t j = 0; j < 2; j ++) {
        run_nextstate[j] = (run_nextstate[j] == h_cell[j].state) ? h_cell[j].next_state : -run_nextstate[j];
    }
    
    // loop of ushort matching
    for (int32_t i = 1; run_nextstate[0] > pat_count_reg || run_nextstate[1] > pat_count_reg; i ++) {
        in_pat = input_buf[gid + i];
        #pragma unroll
        for (int32_t j = 0; j < 2; j ++) {
            if (run_nextstate[j] >= pat_count_reg) {
                pat_key[j] = in_pat;
                pos[j] = hash_info[run_nextstate[j]] + pat_key[j];
                h_cell[j] = (pos[j] >= 0 && pos[j] < (int32_t)hash_data_size_reg) ? hash_data[pos[j]]: h_cell[j];
                run_nextstate[j] = (run_nextstate[j] == h_cell[j].state) ? h_cell[j].next_state : -run_nextstate[j];
            }
        }
    }

    // match the last ushort
    #pragma unroll
    for (int32_t j = 0; j < 2; j ++) {
        if (run_nextstate[j] <= 0) {
            pat_key[j] &= 0x00FF;
            run_nextstate[j] = -run_nextstate[j];
            pos[j] = hash_info[run_nextstate[j]] + pat_key[j];
            h_cell[j] = (pos[j] >= 0 && pos[j] < (int32_t)hash_data_size_reg) ? hash_data[pos[j]] : h_cell[j];
            run_nextstate[j] = (run_nextstate[j] == h_cell[j].state) ? h_cell[j].next_state : 0;
        }
    }
    
    // write to output
    #pragma unroll
    for (int32_t i = 0; i < 2; i ++) {
        match_result[i] = (run_nextstate[i] <= pat_count_reg) ? run_nextstate[i] : 0;
    }
    if (*((int32_t *)match_result)) {
        result[gid] = *((int32_t *)match_result);
    }
}

int32_t DPFAC_InitDevice_CUDA(const int count)
{
    int32_t dev_count;
    HANDLE_ERROR( hipGetDeviceCount(&dev_count) );
    if(dev_count == 0) {
        fprintf(stderr, "There is no device.\n");
        return -1;
    }
    if(dev_count < count) {
        fprintf(stderr, "Device out of range.\n");
        return -1;
    }
    
    HANDLE_ERROR( hipSetDevice(count) );
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    
    return 0;
}

int32_t DPFAC_CompareToPFACFormat_CUDA(const uint16_t *input_buf,     const uint32_t input_buf_size,
                                  const  int32_t *hash_info_buf, const uint32_t hash_info_buf_size,
                                  const  int32_t *hash_data_buf, const uint32_t hash_data_buf_size,
                                  const  int16_t *pat_mask_buf,  const uint32_t pat_mask_buf_size,
                                        uint16_t *output_buf,
                                  const uint16_t  pat_count,     const uint16_t max_pat_block_len)
{
    uint16_t *dev_input_buf;
    int32_t  *dev_hash_info_buf;
    int32_t  *dev_hash_data_buf;
    int16_t  *dev_pat_mask_buf;
    int32_t *dev_output_buf;
    HANDLE_ERROR( hipMalloc(&dev_input_buf, sizeof(uint16_t) * input_buf_size) );
    HANDLE_ERROR( hipMalloc(&dev_hash_info_buf, sizeof(int32_t) * 65536) );
    HANDLE_ERROR( hipMalloc(&dev_hash_data_buf, sizeof(int32_t) * hash_data_buf_size) );
    //HandleError( hipMalloc(&dev_pat_mask_buf, sizeof(int16_t) * pat_mask_buf_size) );
    HANDLE_ERROR( hipMalloc(&dev_output_buf, sizeof(uint16_t) * input_buf_size * 2) );
    
    HANDLE_ERROR( hipMemcpy(dev_input_buf, input_buf, sizeof(uint16_t) * input_buf_size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_output_buf, output_buf, sizeof(uint16_t) * 256 * 1024 * 1024, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_hash_info_buf, hash_info_buf, sizeof(int32_t) * hash_info_buf_size, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(dev_hash_data_buf, hash_data_buf, sizeof(int32_t) * hash_data_buf_size, hipMemcpyHostToDevice) );
    //HandleError( hipMemcpy(dev_pat_mask_buf, pat_mask_buf, sizeof(int16_t) * pat_mask_buf_size, hipMemcpyHostToDevice) );
    
    static float elapsedTime1 = 0.0;
    float elapsedTime;
    static int the_time = 0;
    for (int i =0;i<256;i++) {
        //printf("GPU i=%d\n",i);
    hipEvent_t ev_begin, ev_end;
    hipEventCreate(&ev_begin);
    hipEventCreate(&ev_end);
    hipEventRecord(ev_begin, 0);
    
    // kkkkkkk
   // TestCU<<<65536, 1024>>>(dev_output_buf);
   //dim3 numBlocks(128*1024);
   
    DpfacKernelCU_Normal<<<128 * 1024 * 8, 128>>>(dev_output_buf, dev_input_buf, 256 * 1024 * 1024, dev_hash_info_buf, (HashCell *)dev_hash_data_buf, hash_data_buf_size, pat_count, max_pat_block_len);
    hipError_t err = hipGetLastError();
    //if (err != hipSuccess)
    //    printf("Error: %s\n", hipGetErrorString(err));

    hipEventRecord(ev_end, 0);
    hipEventSynchronize(ev_end);
    
    the_time += 1;
    HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, ev_begin, ev_end) );
    elapsedTime1 += elapsedTime;
    
    hipEventDestroy(ev_begin);
    hipEventDestroy(ev_end);
    
    HANDLE_ERROR( hipDeviceSynchronize() );
    }
    fprintf(stdout, "GPU average time: %lf ms\n", elapsedTime1 / the_time);
    HANDLE_ERROR( hipMemcpy(output_buf, dev_output_buf, sizeof(uint16_t) * 256 * 1024 * 1024, hipMemcpyDeviceToHost) );
    
    HANDLE_ERROR( hipFree(dev_input_buf) );
    HANDLE_ERROR( hipFree(dev_hash_info_buf) );
    HANDLE_ERROR( hipFree(dev_hash_data_buf) );
    //HandleError( hipFree(dev_pat_mask_buf) );
    HANDLE_ERROR( hipFree(dev_output_buf) );
    
    

    return 0;
}
